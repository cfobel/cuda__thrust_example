#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/reduce.h>
#include <thrust/sort.h>
#include <thrust/functional.h>
#include <vector>
#include <iostream>
#include <cstdlib>
#include "mapped_allocator/mapped_allocator.hpp"

int main(void) {
    // generate random data on the host
    const int N = 100;
    thrust::host_vector<int> h_vec(N);
    thrust::generate(h_vec.begin(), h_vec.end(), rand);

    // transfer to device and compute sum
    thrust::device_vector<int> d_vec = h_vec;
    int x = thrust::reduce(d_vec.begin(), d_vec.end(), 0, thrust::plus<int>());

    std::vector<int, mapped_allocator<int> > h_mapped_vec(N); // host mapped memory vector
    int *d_ptr; // pointer to mapped device memory
    thrust::generate(h_mapped_vec.begin(), h_mapped_vec.end(), rand);
    hipHostGetDevicePointer((void **)&d_ptr, (void *)&h_mapped_vec[0], 0);
    thrust::device_ptr<int> d_vec_ptr(d_ptr);
    thrust::sort(d_ptr, d_ptr + N);

    for(int i = 0; i < N; i++) {
        std::cout << d_ptr[i] << std::endl;
    }

    return 0;
}
